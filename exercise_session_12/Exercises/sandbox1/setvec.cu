#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define BLOCKSIZE 256

__global__ void kern_set_val (float *gpu_ptr, float value, int n) {
  int i;
  i = blockIdx.x *blockDim.x + threadIdx.x;
  gpu_ptr[i] = value;
}

int main () {
  int i, failed=0;
  int N = 1024;   // size of vector
  float *ptr;     // Host pointer 
  float *gpu_ptr; // Device pointer
  int size = N * sizeof(float);
  
  /* Allocate vector in Host*/
  ptr = (float *)malloc(sizeof(float)*N);
  /* Allocate vector in Device*/
  hipMalloc (&gpu_ptr, sizeof(float)*N);

  kern_set_val<<<(N+BLOCKSIZE-1)/BLOCKSIZE, BLOCKSIZE>>>(gpu_ptr, 11.0, N);

  hipDeviceSynchronize ();

  hipMemcpy(ptr, gpu_ptr, size, hipMemcpyDeviceToHost);

  hipFree (gpu_ptr);

  /* Now check that it did what we want */

  for (i = 0; i < 10; i++)//first ten values are written
    printf ("%f\t", ptr[i]);
  printf ("\n");
  for (i = N-10; i < N; i++)//last ten values are written
    printf ("%f\t", ptr[i]);
  printf ("\n");

  for (i = 0; i < N; i++) {//All values are compared
    if (fabs(ptr[i]-11.0) > 1e-8) {
      failed=1;
    }
  }
  if (failed) {
    printf ("FAILED !!\n");
  } else {
    printf ("PASSED !!\n");
  }    
  free (ptr);
}

